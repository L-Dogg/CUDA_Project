#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <bitset>
#include <cstdlib>
#include <stdint.h>
#include <time.h>

#define SEQUENCE_LENGTH 262144 // (In bits, 64 multiples only)
#define SEQUENCES_COUNT 256
#define UINT64_CAPACITY 64

#define BLOCK_SIZE 512

const uint64_t m1 = 0x55; //(01)b
const uint64_t m2 = 0x33; //(0011)b
const uint64_t m4 = 0x0f; //(00001111)b

__host__ __device__ int bitCount(uint64_t x);
__global__ void reduce(uint64_t* input, int* output, int len);
__global__ void compareSequences(int n, const uint64_t *a, const uint64_t *b, int* distance, const int distance_idx);

void generateSequences(uint64_t** seq, int uints_required, int** reduce_output, int blocks_count);
int calculateUsingCuda(uint64_t** seq, uint64_t** dev_seq,
	int* distances, int * dev_distances,
	int** reduce_output, int** dev_reduce_output, int reduce_output_length,
	int uints_required, int blocks_count);
int countPairs(int* distance);
void calculateUsingCPU(uint64_t** seq, int uints_required, int* distances);

int main(int argc, char ** argv)
{
	int uints_required = SEQUENCE_LENGTH / UINT64_CAPACITY;
	int blocks_count = (uints_required + BLOCK_SIZE - 1) / BLOCK_SIZE;
	int reduce_output_length = uints_required / (BLOCK_SIZE << 1);

	uint64_t **seq = nullptr, **dev_seq = nullptr;
	int *distances = nullptr, *dev_distances = nullptr;
	int **reduce_output = nullptr, **dev_reduce_output = nullptr;
	int *sum = nullptr;

	seq = (uint64_t **)malloc(SEQUENCES_COUNT * sizeof(uint64_t *));
	dev_seq = (uint64_t **)malloc(SEQUENCES_COUNT * sizeof(uint64_t *));

	reduce_output = (int **)malloc(SEQUENCES_COUNT * sizeof(int *));
	dev_reduce_output = (int **)malloc(SEQUENCES_COUNT * sizeof(int *));

	distances = (int *)malloc(SEQUENCES_COUNT * SEQUENCES_COUNT * sizeof(int));

	printf("Generating sequences.\n");
	clock_t begin = clock();
	generateSequences(seq, uints_required, reduce_output, blocks_count);
	clock_t end = clock();

	double elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
	printf("Generated sequences. Time elapsed %fs.\n", elapsed_secs);

	printf("Starting CUDA computation.\n");
	begin = clock();
	int ret = calculateUsingCuda(seq, dev_seq, distances, dev_distances,
		reduce_output, dev_reduce_output, reduce_output_length, uints_required, blocks_count);
	end = clock();

	if (ret != 0)
		return -1;

	elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
	int pairs = countPairs(distances);
	printf("Found %d pairs. Time elapsed %fs.\n", pairs, elapsed_secs);

	hipFree(dev_seq);
	hipFree(dev_distances);
	hipDeviceReset();

	memset(distances, 0, SEQUENCES_COUNT * SEQUENCES_COUNT);

	printf("Starting CPU computation.\n");
	begin = clock();
	calculateUsingCPU(seq, uints_required, distances);
	end = clock();

	pairs = countPairs(distances);
	elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
	printf("Found %d pairs. Time elapsed %fs.\n", pairs, elapsed_secs);

	return 0;
}

void generateSequences(uint64_t** seq, int uints_required, int** reduce_output, int blocks_count)
{
	seq[0] = (uint64_t *)malloc(uints_required * sizeof(uint64_t));
	reduce_output[0] = (int*)malloc(sizeof(int) * blocks_count);
	seq[1] = (uint64_t *)malloc(uints_required * sizeof(uint64_t));
	reduce_output[1] = (int*)malloc(sizeof(int) * blocks_count);
	for (int j = 0; j < uints_required; j++)
		for (int k = 0; k <= 63; k++)
		{
			uint64_t val = (uint64_t)(rand() % 2) << k;
			seq[0][j] += val;
			seq[1][j] += val;
		}

	if (seq[0][0] & 0x1)
		seq[1][0] -= 1;
	else
		seq[1][0] += 1;

	for (int i = 2; i < SEQUENCES_COUNT; i++)
	{
		srand(2137 + i);
		seq[i] = (uint64_t *)malloc(uints_required * sizeof(uint64_t));
		reduce_output[i] = (int*)malloc(sizeof(int) * blocks_count);
		for (int j = 0; j < uints_required; j++)
			for (int k = 0; k <= 63; k++)
				seq[i][j] += (uint64_t)(rand() % 2) << k;
	}
}

int countPairs(int* distances)
{
	int pairs = 0;

	for (int i = 0; i < SEQUENCES_COUNT; i++)
		for (int j = i + 1; j < SEQUENCES_COUNT; j++)
			if (distances[j + i * SEQUENCES_COUNT] == 1)
				pairs++;

	return pairs;
}

int calculateUsingCuda(uint64_t** seq, uint64_t** dev_seq,
	int* distances, int* dev_distances,
	int** reduce_output, int** dev_reduce_output, int reduce_output_length,
	int uints_required, int blocks_count)
{
	int *sums;
	hipError_t cudaStatus;

	cudaStatus = hipMalloc(&dev_distances, SEQUENCES_COUNT * SEQUENCES_COUNT * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemset(dev_distances, 0, SEQUENCES_COUNT * SEQUENCES_COUNT * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemset failed!");
		goto Error;
	}

	for (int i = 0; i < SEQUENCES_COUNT; i++)
	{
		cudaStatus = hipMalloc(&dev_seq[i], uints_required * sizeof(uint64_t));
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		cudaStatus = hipMemcpy(dev_seq[i], seq[i], uints_required * sizeof(uint64_t), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		cudaStatus = hipMalloc(&dev_reduce_output[i], sizeof(int) * reduce_output_length);
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		hipMemset(dev_reduce_output[i], 0, sizeof(int) * reduce_output_length);
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMemset failed!");
			goto Error;
		}
	}

	for (int i = 0; i < SEQUENCES_COUNT; i++)
	{
		reduce << <reduce_output_length, BLOCK_SIZE >> > (dev_seq[i], dev_reduce_output[i], uints_required);
	}

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Couldn't launch reduce: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	sums = (int *)malloc(SEQUENCES_COUNT * sizeof(int));
	memset(sums, 0, SEQUENCES_COUNT);

	for (int i = 0; i < SEQUENCES_COUNT; i++)
	{
		cudaStatus = hipMemcpy(reduce_output[i], dev_reduce_output[i], reduce_output_length * sizeof(int), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMemcpy failed! - reduce output\n");
			fprintf(stderr, "i = %d, status = %s\n", i, hipGetErrorString(cudaStatus));
			if (reduce_output[i] == nullptr)
				fprintf(stderr, "reduce_output null\n");
			if (reduce_output[i] == nullptr)
				fprintf(stderr, "dev_reduce_output null");
			goto Error;
		}
		for (int j = 0; j < reduce_output_length; j++)
			sums[i] += reduce_output[i][j];
	}

	for (int i = 0; i < SEQUENCES_COUNT; i++)
	{
		for (int j = i + 1; j < SEQUENCES_COUNT; j++)
		{
			if (sums[i] - sums[j] == 1 || sums[i] - sums[j] == -1)
			{
				compareSequences << <blocks_count, BLOCK_SIZE >> > (uints_required, dev_seq[i], dev_seq[j], dev_distances, j + SEQUENCES_COUNT*i);

				cudaStatus = hipGetLastError();
				if (cudaStatus != hipSuccess)
				{
					fprintf(stderr, "Couldn't launch compareSequences: %s\n", hipGetErrorString(cudaStatus));
					goto Error;
				}
			}
		}
	}

	cudaStatus = hipMemcpy(distances, dev_distances, SEQUENCES_COUNT * SEQUENCES_COUNT * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed! - distances");
		goto Error;
	}

	return 0;

Error:
	hipFree(dev_seq);
	free(seq);
	hipFree(dev_distances);
	free(distances);
	hipDeviceReset();

	return -1;
}

void calculateUsingCPU(uint64_t** seq, int uints_required, int* distances)
{
	for (int i = 0; i < SEQUENCES_COUNT; i++)
		for (int j = i + 1; j < SEQUENCES_COUNT; j++)
			for (int k = 0; k < uints_required; k++)
				distances[j + SEQUENCES_COUNT * i] += bitCount(seq[i][k] ^ seq[j][k]);
}

__global__ void compareSequences(int n, const uint64_t *a, const uint64_t *b, int* distance, const int distance_idx)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < n && a[index] != b[index] && distance[distance_idx] < 2)
	{
		atomicAdd(&distance[distance_idx], bitCount(a[index] ^ b[index]));
	}
}

/*
* Source:
*	http://stackoverflow.com/questions/109023/how-to-count-the-number-of-set-bits-in-a-32-bit-integer
*/
__host__ __device__ int bitCount(uint64_t x)
{
	x -= (x >> 1) & m1;
	x = (x & m2) + ((x >> 2) & m2);
	x = (x + (x >> 4)) & m4;
	x += x >> 8;
	x += x >> 16;
	x += x >> 32;

	return x & 0x7f;
}

__global__ void reduce(uint64_t* input, int* output, int len)
{
	//Load a segment of the input vector into shared memory
	__shared__ unsigned int partialSum[2 * BLOCK_SIZE];
	unsigned int t = threadIdx.x, start = 2 * blockIdx.x * BLOCK_SIZE;

	if (start + t < len)
		partialSum[t] = bitCount(input[start + t]);
	else
		partialSum[t] = 0;
	if (start + BLOCK_SIZE + t < len)
		partialSum[BLOCK_SIZE + t] = bitCount(input[start + BLOCK_SIZE + t]);
	else
		partialSum[BLOCK_SIZE + t] = 0;

	// Traverse the reduction tree
	for (unsigned int stride = BLOCK_SIZE; stride >= 1; stride >>= 1) {
		__syncthreads();
		if (t < stride)
			partialSum[t] += partialSum[t + stride];
	}

	// Write the computed sum of the block to the output vector at the correct index
	if (t == 0)
		output[blockIdx.x] = partialSum[0];
}